#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2019-present, Husky Data Lab.
 * All rights reserved.
 *
 * This source code is licensed under the BSD+Patents license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "HQSecondStage.cuh"

#include "../GpuResources.h"
#include "PQCodeDistances.cuh"
#include "PQCodeLoad.cuh"
#include "../utils/ConversionOperators.cuh"
#include "../utils/DeviceTensor.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/Float16.cuh"
#include "../utils/LoadStoreOperators.cuh"
#include "../utils/StaticUtils.h"
#include "../utils/ThrustAllocator.cuh"
#include "IVFUtils.cuh"
#include "LoadCodeDistances.cuh"

#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform_scan.h>

namespace faiss { namespace gpu {

/*template <typename ListIdT = unsigned long long>
__global__ void HQCalcListIds(const Tensor<int, 3, true> imiIndices, const Tensor<int, 2, true> imiUpperBounds, int nprobeSquareLen, int imiSize, Tensor<ListIdT, 2, true> listIds) {
    int qid = blockIdx.x;
    int overallRank = threadIdx.x;

    int imiUpperBoundCol = imiUpperBounds[1][qid];

    int upperBlockSize = imiUpperBoundCol * nprobeSquareLen;

    int coarseRank0, coarseRank1;
    if (overallRank > upperBlockSize) {
        coarseRank0 = nprobeSquareLen + (overallRank - upperBlockSize) / nprobeSquareLen;
        coarseRank1 = (overallRank - upperBlockSize) % nprobeSquareLen;
    } else {
        coarseRank0 = overallRank / imiUpperBoundCol;
        coarseRank1 = overallRank % imiUpperBoundCol;
    }

    listIds[qid][overallRank] = (ListIdT)imiIndices[0][qid][coarseRank0] * imiSize + (ListIdT)imiIndices[1][qid][coarseRank1];
}*/

template <typename ListIdT = unsigned long long>
void runHQCalcListIds(const Tensor<int, 3, true>& deviceIMIIndices, const Tensor<int, 2, true>& deviceIMIUpperBounds, int numQueries, int numListsPerQuery, int nprobeSquareLen, int imiSize, Tensor<ListIdT, 2, true>& deviceListIds, hipStream_t stream) {
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last = first + numQueries * numListsPerQuery;

    // TODO: should I make a class instead of a lambda? I can control how to store the captured variables if I use a class.
    auto pos2ListId = [=] __device__ (int pos) {
        int qid = pos / numListsPerQuery;
        int overallRank = pos % numListsPerQuery;

        int imiUpperBoundCol = deviceIMIUpperBounds[1][qid];

        int upperBlockSize = imiUpperBoundCol * nprobeSquareLen;

        int coarseRank0, coarseRank1;
        if (overallRank > upperBlockSize) {
            coarseRank0 = nprobeSquareLen + (overallRank - upperBlockSize) / nprobeSquareLen;
            coarseRank1 = (overallRank - upperBlockSize) % nprobeSquareLen;
        } else {
            coarseRank0 = overallRank / imiUpperBoundCol;
            coarseRank1 = overallRank % imiUpperBoundCol;
        }

        return (ListIdT)deviceIMIIndices[0][qid][coarseRank0] * imiSize + (ListIdT)deviceIMIIndices[1][qid][coarseRank1];
    };

    thrust::transform(thrust::cuda::par.on(stream), first, last, deviceListIds.data(), pos2ListId);

    //HQCalcListIds<<<numQueries, numListsPerQuery, 0, stream>>>(deviceIMIIndices, deviceIMIUpperBounds, nprobeSquareLen, imiSize, deviceListIds);
}

template <typename ListIdT = unsigned long long>
void runHQCalcListOffsets(const int* deviceListLengths, const Tensor<ListIdT, 2, true>& deviceListIds, int* devicePrefixSumOffsets, GpuResources* resources, hipStream_t stream) {
    constexpr int kThrustMemSize = 16384; // TODO: set a reasonable size

    auto& mem = resources->getMemoryManagerCurrentDevice();

    DeviceTensor<char, 1, true> thrustMem(
      mem, {kThrustMemSize}, stream);

    GpuResourcesThrustAllocator thrustAlloc(thrustMem.data(),
                                            thrustMem.getSizeInBytes());

    // TODO: should I make a class instead of a lambda? I can control how to store the captured variables if I use a class.
    auto listId2ListLength = [=] __device__ (ListIdT listId) {
        return deviceListLengths[listId];
    };

    thrust::transform_inclusive_scan(thrust::cuda::par(thrustAlloc).on(stream), deviceListIds.data(), deviceListIds.end(), devicePrefixSumOffsets, listId2ListLength, thrust::plus<int>());
}

template <typename LookupVecT, typename ListIdT, typename LookupT>
__global__ void
HQSecondStageDistances(// (qid, overallRank) -> listId
                       const Tensor<ListIdT, 2, true> listIds,
                       // (imiId, qid) -> upper_bound
                       const Tensor<int, 2, true> imiUpperBounds,
                       // (imiId, qid, coarseRank, fineIdx) -> val
                       const Tensor<LookupT, 4, true> distanceTable,
                       // (listId, i) -> item
                       // Should contains only the codes related to this function. Other codes (those for the third stage) should be stored separately
                       const void** listCodes,
                       // listId -> len
                       const int* listLengths,
                       // (qid, overallRank) -> offset
                       const Tensor<int, 2, true> prefixSumOffsets,
                       int nprobeSquareLen,
                       int imiSize,
                       // offset -> distance
                       Tensor<float, 1, true> distances) {
  constexpr int NumSubQuantizers = 2; // TODO: extend to possibly more sub quantizers

  // Where the pq code -> residual distance is stored
  extern __shared__ char smemCodeDistances[];
  LookupT* codeDist0 = (LookupT*) smemCodeDistances;
  LookupT* codeDist1 = (LookupT*) smemCodeDistances + distanceTable.getSize(3);

  // Each block handles a single bucket
  int qid = blockIdx.y;
  int overallRank = blockIdx.x;

  int imiUpperBoundCol = imiUpperBounds[1][qid];

  int upperBlockSize = imiUpperBoundCol * nprobeSquareLen;
  int coarseRank0, coarseRank1;
  if (overallRank > upperBlockSize) {
    coarseRank0 = nprobeSquareLen + (overallRank - upperBlockSize) / nprobeSquareLen;
    coarseRank1 = (overallRank - upperBlockSize) % nprobeSquareLen;
  } else {
    coarseRank0 = overallRank / imiUpperBoundCol;
    coarseRank1 = overallRank % imiUpperBoundCol;
  }

  // This is where we start writing out data
  // We ensure that before the array (at offset -1), there is a 0 value
  int outBase = *(prefixSumOffsets[qid][overallRank].data() - 1);
  float* distanceOut = distances[outBase].data();

  ListIdT listId = listIds[qid][overallRank];
  // Safety guard in case NaNs in input cause no list ID to be generated
  if (listId == (ListIdT)-1) {
    return;
  }

  unsigned char* codeList = (unsigned char*) listCodes[listId];
  int limit = listLengths[listId];

  constexpr int kNumCode32 = NumSubQuantizers <= 4 ? 1 :
    (NumSubQuantizers / 4);
  unsigned int code32[kNumCode32];
  unsigned int nextCode32[kNumCode32];

  // We double-buffer the code loading, which improves memory utilization
  if (threadIdx.x < limit) {
    LoadCode32<NumSubQuantizers>::load(code32, codeList, threadIdx.x);
  }

  LoadCodeDistances<LookupT, LookupVecT>::load(
    codeDist0,
    distanceTable[0][qid][coarseRank0].data(),
    distanceTable.getSize(3));
  LoadCodeDistances<LookupT, LookupVecT>::load(
    codeDist1,
    distanceTable[1][qid][coarseRank1].data(),
    distanceTable.getSize(3));

  // Prevent WAR dependencies
  __syncthreads();

  // Each thread handles one code element in the list, with a
  // block-wide stride
  for (int codeIndex = threadIdx.x;
       codeIndex < limit;
       codeIndex += blockDim.x) {
    // Prefetch next codes
    if (codeIndex + blockDim.x < limit) {
      LoadCode32<NumSubQuantizers>::load(
        nextCode32, codeList, codeIndex + blockDim.x);
    }

    float dist = 0.0f;

#pragma unroll
    for (int word = 0; word < kNumCode32; ++word) {
      auto code = getByte(code32[word], 0, 8);
      dist += ConvertTo<float>::to(codeDist0[code]);

      code = getByte(code32[word], 8, 8);
      dist += ConvertTo<float>::to(codeDist1[code]);
    }

    // Write out intermediate distance result
    // We do not maintain indices here, in order to reduce global
    // memory traffic. Those are recovered in the final selection step.
    distanceOut[codeIndex] = dist;

    // Rotate buffers
#pragma unroll
    for (int word = 0; word < kNumCode32; ++word) {
      code32[word] = nextCode32[word];
    }
  }
}

template <typename ListIdT, typename LookupT>
void runHQSecondStageDistances(// (qid, overallRank) -> listId
                               const Tensor<ListIdT, 2, true>& deviceListIds,
                               // (imiId, qid) -> upper_bound
                               const Tensor<int, 2, true>& deviceIMIUpperBounds,
                               // (imiId, qid, coarseRank, fineIdx) -> val
                               const Tensor<LookupT, 4, true>& deviceDistanceTable,
                               // (listId, i) -> item
                               // Should contains only the codes related to this function. Other codes (those for the third stage) should be stored separately
                               const void** deviceListCodes,
                               // listId -> len
                               const int* deviceListLengths,
                               // (qid, overallRank) -> offset
                               const Tensor<int, 2, true>& devicePrefixSumOffsets,
                               int numQueries,
                               int nprobeSquareLen,
                               int imiSize,
                               // offset -> distance
                               Tensor<float, 1, true>& deviceDistances,
                               hipStream_t stream) {
  auto kThreadsPerBlock = 64;

  auto grid = dim3(devicePrefixSumOffsets.getSize(1), numQueries);
  auto block = dim3(kThreadsPerBlock);

  // pq centroid distances
  auto smem = sizeof(LookupT);
  smem *= deviceDistanceTable.getSize(3) * deviceDistanceTable.getSize(0);
  FAISS_ASSERT(smem <= getMaxSharedMemPerBlockCurrentDevice());

#define RUN_PQ_OPT(LOOKUP_VEC_T)                            \
  do {                                                      \
    HQSecondStageDistances<LOOKUP_VEC_T>                    \
      <<<grid, block, smem, stream>>>(                      \
        deviceListIds,                                      \
        deviceIMIUpperBounds,                               \
        deviceDistanceTable,                                \
        deviceListCodes,                                    \
        deviceListLengths,                                  \
        devicePrefixSumOffsets,                             \
        nprobeSquareLen,                                    \
        imiSize,                                            \
        deviceDistances);                                   \
  } while (0)

#ifdef FAISS_USE_FLOAT16
#define RUN_PQ()                       \
  do {                                 \
    if (sizeof(LookupT) == 2) {        \
      RUN_PQ_OPT(Half8);               \
    } else {                           \
      RUN_PQ_OPT(float4);              \
    }                                  \
  } while (0)
#else
#define RUN_PQ()                       \
    do {                               \
      RUN_PQ_OPT(float4);              \
    } while (0)
#endif // FAISS_USE_FLOAT16

  RUN_PQ();

#undef RUN_PQ
#undef RUN_PQ_OPT
}

void runHQSecondStage(const Tensor<int, 3, true>& deviceIMIIndices,
                      const Tensor<int, 2, true>& deviceIMIUpperBounds,
                      const Tensor<float, 4, true>& deviceDistanceTable,
                      const void** deviceListCodes,
                      const int* deviceListLengths,
                      int numQueries,
                      int k,
                      int numListsPerQuery,
                      int nprobeSquareLen,
                      int imiSize,
                      bool chooseLargest,
                      // (field, qid, rank) -> val
                      // field 0: imiId0
                      // field 1: imiId1
                      // field 2: fineId
                      Tensor<int, 3, true>& deviceOutIndices,
                      GpuResources* resources,
                      hipStream_t stream) {
    auto& mem = resources->getMemoryManagerCurrentDevice();

    using ListIdT = unsigned long long; // TODO: make it configurable

    DeviceTensor<ListIdT, 2, true> deviceListIds(mem,
            {numQueries, numListsPerQuery}, stream);
    runHQCalcListIds(deviceIMIIndices,
                     deviceIMIUpperBounds,
                     numQueries,
                     numListsPerQuery,
                     nprobeSquareLen,
                     imiSize,
                     deviceListIds,
                     stream);

    DeviceTensor<int, 2, true> devicePrefixSumOffsets(mem,
            {numQueries, numListsPerQuery}, stream);
    runHQCalcListOffsets(deviceListLengths,
                         deviceListIds,
                         devicePrefixSumOffsets.data(),
                         resources,
                         stream);

    constexpr int maxListLen = 128; // FIXME: set a correct value
    DeviceTensor<float, 2, true> deviceDistances(mem, {numQueries, maxListLen * numListsPerQuery}, stream);
    Tensor<float, 1, true> deviceDistancesFlat = deviceDistances.downcastInner<1>();

    runHQSecondStageDistances(deviceListIds,
                              deviceIMIUpperBounds,
                              deviceDistanceTable,
                              deviceListCodes,
                              deviceListLengths,
                              devicePrefixSumOffsets,
                              numQueries,
                              nprobeSquareLen,
                              imiSize,
                              deviceDistancesFlat,
                              stream);

    constexpr int kNProbeSplit = 8;
    int pass2Chunks = std::min(numListsPerQuery, kNProbeSplit);

    DeviceTensor<float, 3, true> deviceHeapDistances(mem, {numQueries, pass2Chunks, k}, stream);
    DeviceTensor<int, 3, true> deviceHeapIndices(mem, {numQueries, pass2Chunks, k}, stream);

    runPass1SelectLists(devicePrefixSumOffsets,
                        deviceDistancesFlat,
                        numListsPerQuery,
                        k,
                        chooseLargest,
                        deviceHeapDistances,
                        deviceHeapIndices,
                        stream);

    Tensor<float, 2, true> deviceHeapDistancesFlat = deviceHeapDistances.downcastInner<2>();
    Tensor<int, 2, true> deviceHeapIndicesFlat = deviceHeapIndices.downcastInner<2>();
    runPass2SelectIMILists(deviceHeapDistancesFlat,
                           deviceHeapIndicesFlat,
                           devicePrefixSumOffsets,
                           deviceListIds,
                           k,
                           imiSize,
                           chooseLargest,
                           deviceOutIndices,
                           stream);
}

} } // namespace
