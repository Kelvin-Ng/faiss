#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2019-present, Husky Data Lab.
 * All rights reserved.
 *
 * This source code is licensed under the BSD+Patents license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "SimpleIMICut.cuh"

#include "../../FaissAssert.h"
#include "../utils/ConversionOperators.cuh"
#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/MathOperators.cuh"
#include "../utils/PtxUtils.cuh"
#include "../utils/StaticUtils.h"
#include "../utils/Reductions.cuh"

#include <thrust/tuple.h>

#include <cmath>
#include <cstdio>

namespace faiss { namespace gpu {

// Input: (batch x dim), # repeats
// Output: (# repeats, norm of batch vector)
// Done under the presumption that the dimension size is not too large
// (<10k or so), since there wouldn't be enough parallelism applying a
// single block to the problem. Also that each vector is large enough
// (>64), since a single block works on multiple rows' norms at the
// same time.
// T: the type we are doing the math in (e.g., float, half)
// TVec: the potentially vectorized type we are loading in (e.g.,
// float4, half2)
template <typename T, typename TVec, typename int64_t,
          int RowTileSize, bool NormLoop>
__global__ void simpleIMICut(Tensor<TVec, 3, true, int64_t> input,
                             Tensor<int, 2, true, int64_t> output,
                             int squareLen,
                             int totalLen) {
  
  extern __shared__ char smemByte[]; // #warps * RowTileSize elements
  T* __restrict__ smemMin = (T*) smemByte;
  int64_t numWarps = utils::divUp(blockDim.x, kWarpSize);
  int* __restrict__ smemId  = (int*) (smemByte + sizeof(T) * RowTileSize * numWarps);
  int64_t laneId = getLaneId();
  int64_t warpId = threadIdx.x / kWarpSize;

  bool lastRowTile = (blockIdx.x == (gridDim.x - 1));
  int64_t rowStart = RowTileSize * blockIdx.x;
  T rowMin[RowTileSize];
  int rowId[RowTileSize];
  T minVal;
  int minId;
  constexpr int stp = sizeof(TVec) / sizeof(T);
  int startPos = (squareLen - 1) / stp;
  int totalCol = (totalLen - 2 * squareLen + 1) / stp;
  int upperCol = totalLen / stp;
  if (lastRowTile) {
    // We are handling the very end of the input matrix rows
    for (int64_t row = 0; row < input.getSize(1) - rowStart; ++row) {
      rowMin[0] = 1e100;
      if (NormLoop) {
        for (int64_t col = threadIdx.x;
             col < totalCol; col += blockDim.x) {
          TVec val = Math<TVec>::abs(Math<TVec>::revSub(input[0][rowStart + row][startPos + col], input[1][rowStart + row][upperCol - (startPos + col) - 2]));
          minId = Math<TVec>::argMin(val);
          minVal = Math<TVec>::getVal(val, minId);

          rowId[0] = (minVal < rowMin[0] ? (startPos + col) * stp + minId : rowId[0]);
          rowMin[0] = min(minVal, rowMin[0]);
        }
      } else {
        if (threadIdx.x < totalCol) {
          TVec val = Math<TVec>::abs(Math<TVec>::revSub(input[0][rowStart + row][startPos + threadIdx.x], input[1][rowStart + row][upperCol - (startPos + threadIdx.x) - 2]));
          minId = Math<TVec>::argMin(val);
          minVal = Math<TVec>::getVal(val, minId);

          rowId[0] = (minVal < rowMin[0] ? (startPos + threadIdx.x) * stp + minId : rowId[0]);
          rowMin[0] = min(minVal, rowMin[0]);
        }
      }
      thrust::tie(rowMin[0], rowId[0]) = warpReduceAllMin(rowMin[0], rowId[0]);
      if (laneId == 0) {
        smemMin[row * numWarps + warpId] = rowMin[0];
        smemId[row * numWarps + warpId] = rowId[0];
      }
    }
  } else {
    // We are guaranteed that all RowTileSize rows are available in
    // [rowStart, rowStart + RowTileSize)

#pragma unroll
    for (int row = 0; row < RowTileSize; ++row) {
      rowMin[row] = 1e100;
      rowId[row] = -123;
    }

    if (NormLoop) {
      // A single block of threads is not big enough to span each
      // vector
      TVec tmp[RowTileSize];

      for (int64_t col = threadIdx.x;
           col < totalCol; col += blockDim.x) {

#pragma unroll
        for (int row = 0; row < RowTileSize; ++row) {
          tmp[row] = Math<TVec>::abs(Math<TVec>::revSub(input[0][rowStart + row][startPos + col], input[1][rowStart + row][upperCol - (startPos + col) - 2]));
        }

#pragma unroll
        for (int row = 0; row < RowTileSize; ++row) {
          minId = Math<TVec>::argMin(tmp[row]);
          minVal = Math<TVec>::getVal(tmp[row], minId);
          rowId[row] = (minVal < rowMin[row] ? (startPos + col) * stp + minId : rowId[row]);
          rowMin[row] = min(minVal, rowMin[row]);
        }   
      }
    } else {
      if (threadIdx.x < totalCol) {
        TVec tmp[RowTileSize];

        // A block of threads is the exact size of the vector
#pragma unroll
        for (int row = 0; row < RowTileSize; ++row) {
          tmp[row] = Math<TVec>::abs(Math<TVec>::revSub(input[0][rowStart + row][startPos + threadIdx.x], input[1][rowStart + row][upperCol - (startPos + threadIdx.x) - 2]));
        }

#pragma unroll
        for (int row = 0; row < RowTileSize; ++row) {
          minId = Math<TVec>::argMin(tmp[row]);
          minVal = Math<TVec>::getVal(tmp[row], minId);
          rowId[row] = (minVal < rowMin[row] ? (startPos + threadIdx.x) * stp + minId : rowId[row]);
          rowMin[row] = min(minVal, rowMin[row]);
        }
      }
    }

    // Sum up all parts in each warp
#pragma unroll
    for (int row = 0; row < RowTileSize; ++row) {
      thrust::tie(rowMin[row], rowId[row]) = warpReduceAllMin(rowMin[row], rowId[row]);
    }

    if (laneId == 0) {
#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        smemMin[row * numWarps + warpId] = rowMin[row];
        smemId[row * numWarps + warpId] = rowId[row];
      }
    }
  }

  __syncthreads();

  // Sum across warps
  if (warpId == 0) {
#pragma unroll
    for (int row = 0; row < RowTileSize; ++row) {
      rowMin[row] = laneId < numWarps ? smemMin[row * numWarps + laneId] : 1e100;
      rowId[row] = laneId < numWarps ? smemId[row * numWarps + laneId] : Math<T>::zero();
    }

#pragma unroll
    for (int row = 0; row < RowTileSize; ++row) {
      thrust::tie(rowMin[row], rowId[row]) = warpReduceAllMin(rowMin[row], rowId[row]);
    }

    // Write out answer
    if (laneId == 0) {
#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        int outCol = rowStart + row;
        if (lastRowTile) {
          if (outCol < output.getSize(1)) {
            output[0][outCol] = rowId[row];
            output[1][outCol] = totalLen - rowId[row] - 2;
          }
        } else {
          output[0][outCol] = rowId[row];
          output[1][outCol] = totalLen - rowId[row] - 2;
        }
      }
    }
  }
}

template <typename T, typename TVec, typename int64_t>
void runSimpleIMICut(const Tensor<T, 3, true, int64_t>& input,
                     Tensor<int, 2, true, int64_t>& output,
                     int squareLen,
                     int totalLen,
                     hipStream_t stream) {
  FAISS_ASSERT(input.getSize(0) == output.getSize(0));

  int64_t maxThreads = (int64_t) getMaxThreadsCurrentDevice();

  constexpr int rowTileSize = 8;

#define RUN_L2(TYPE_T, TYPE_TVEC, INPUT)                                            \
  do {                                                                              \
    if (normLoop) {                                                                 \
      simpleIMICut<TYPE_T, TYPE_TVEC, int64_t, rowTileSize, true>                   \
        <<<grid, block, smem, stream>>>(INPUT, output, squareLen, totalLen);        \
    } else {                                                                        \
      simpleIMICut<TYPE_T, TYPE_TVEC, int64_t, rowTileSize, false>                  \
        <<<grid, block, smem, stream>>>(INPUT, output, squareLen, totalLen);        \
    }                                                                               \
  }while (0)                                                                
          
  // Make sure that the considered segment [squareLen - 1, totalLen - squareLen + 1] located in some complete pieces of float4,
  // which requested that (squareLen - 1) % 4 == 0 and (totalLen - 2 * squareLen + 1) % 4 == 0
  // equals to squareLen % 4 == 1 and totalLen % 4 == 3
  if (input.template canCastResize<TVec>() && (squareLen % 4 == 1) && (totalLen % 4 == 1)) {

    // Can load using the vectorized type
    auto inputV = input.template castResize<TVec>();

    auto dim = (totalLen - 2 * squareLen + 1) / 4;
    bool normLoop = dim > maxThreads;
    auto numThreads = min(dim, (int)maxThreads);
    if (numThreads % 32 != 0) {
      numThreads += (32 - numThreads % 32);
    }

    auto grid = dim3(utils::divUp(inputV.getSize(1), rowTileSize));
    auto block = dim3(numThreads);

    auto smem = (sizeof(T) + sizeof(int)) * rowTileSize * utils::divUp(numThreads, kWarpSize);

    RUN_L2(T, TVec, inputV);
  } else {
    // Can't load using the vectorized type
    auto dim = totalLen - 2 * squareLen + 1;
    bool normLoop = dim > maxThreads;
    auto numThreads = min(dim, (int)maxThreads);
    if (numThreads % 32 != 0) {
      numThreads += (32 - numThreads % 32);
    }

    auto grid = dim3(utils::divUp(input.getSize(1), rowTileSize));
    auto block = dim3(numThreads);

    auto smem = (sizeof(T) + sizeof(int)) * rowTileSize * utils::divUp(numThreads, kWarpSize);
    RUN_L2(T, T, input);
  }
#undef RUN_L2

  CUDA_TEST_ERROR();

}

void runSimpleIMICut(const Tensor<float, 3, true>& input,
                     Tensor<int, 2, true>& output,
                     int squareLen,
                     int totalLen,
                     hipStream_t stream) {
  if (input.canUseIndexType<int>()) {
    runSimpleIMICut<float, float4, int>(input, output, squareLen, totalLen, stream);
  } else {
    auto inputCast = input.castIndexType<long>();
    auto outputCast = output.castIndexType<long>();
    runSimpleIMICut<float, float4, long>(inputCast, outputCast, squareLen, totalLen, stream);
  }
}

} } // namespace
