#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2019-present, Husky Data Lab.
 * All rights reserved.
 *
 * This source code is licensed under the BSD+Patents license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "HQThirdStageL2Distances.cuh"
#include "../../FaissAssert.h"
#include "../utils/ConversionOperators.cuh"
#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/Float16.cuh"
#include "../utils/MathOperators.cuh"
#include "../utils/PtxUtils.cuh"
#include "../utils/StaticUtils.h"
#include "../utils/Reductions.cuh"

namespace faiss { namespace gpu {

__device__ __forceinline__ void
printCodewords2(int a, int b, int c, int d, float4 val) {
    printf("codewords2[%d][%d][%d][%d]: (%f %f %f %f)\n", a, b, c, d, val.x, val.y, val.z, val.w);
}

__device__ __forceinline__ void
printCodewords2(int a, int b, int c, int d, float val) {
    printf("codewords2[%d][%d][%d][%d]: %f\n", a, b, c, d, val);
}

__device__ __forceinline__ void
printCodewords1(int a, int b, int c, int d, float4 val) {
    printf("codewords1[%d][%d][%d][%d]: (%f %f %f %f)\n", a, b, c, d, val.x, val.y, val.z, val.w);
}

__device__ __forceinline__ void
printCodewords1(int a, int b, int c, int d, float val) {
    printf("codewords1[%d][%d][%d][%d]: %f\n", a, b, c, d, val);
}

template <int numCodes2, typename T, typename TVec>
__device__ __forceinline__ T
HQThirdStageL2DistancesOneCol(const Tensor<TVec, 2, true>& queries,
                              const void** listCodes1,
                              const void** listCodes2,
                              const Tensor<TVec, 4, true>& codewords1,
                              const Tensor<TVec, 4, true>& codewords2,
                              int imiId[2],
                              int fineId,
                              int listId,
                              int col) {
  int halfDim = queries.getSize(1) / 2;
  bool isSecondHalf = (col >= halfDim);

  const unsigned char* myListCodes1 = (const unsigned char*)listCodes1[listId];
  const unsigned char* myListCodes2 = (const unsigned char*)listCodes2[listId];

  unsigned char code1 = myListCodes1[fineId * 2 + isSecondHalf];
  unsigned char codes2[numCodes2];

#pragma unroll
  for (int i = 0; i < numCodes2; ++i) {
    codes2[i] = myListCodes2[fineId * numCodes2 + i];
  }

  TVec val = queries[blockIdx.y][col];

  TVec my_codewords[1 + numCodes2];

  my_codewords[0] = codewords1[isSecondHalf][imiId[isSecondHalf]][code1][col - isSecondHalf * halfDim];
  if (imiId[0] == 3353 && imiId[1] == 1055) {
      printCodewords1(isSecondHalf, imiId[isSecondHalf], code1, col - isSecondHalf * halfDim, my_codewords[0]);
  }
#pragma unroll
  for (int i = 0; i < numCodes2 / 2; ++i) {
    my_codewords[1 + i * 2 + 0] = codewords2[i][0                    ][codes2[i * 2 + 0]][col];
    my_codewords[1 + i * 2 + 1] = codewords2[i][1 + codes2[i * 2 + 0]][codes2[i * 2 + 1]][col];
    //printCodewords2(i, 0, codes2[i * 2 + 0], col, my_codewords[1 + i * 2 + 0]);
    //printCodewords2(i, 1 + codes2[i * 2 + 0], codes2[i * 2 + 1], col, my_codewords[1 + i * 2 + 1]);
  }

#pragma unroll
  for (int i = 0; i < 1 + numCodes2; ++i) {
    val = Math<TVec>::sub(val, my_codewords[i]);
  }
  val = Math<TVec>::mul(val, val);

  return Math<TVec>::reduceAdd(val);
}

// TODO:
// 1. When looping along rows, use grid-stride loops
// 2. Load the codes for all rows before computation to allow more memory transactions in flight
template <typename T, typename TVec,
          int RowTileSize, bool NormLoop, bool NormSquared, int numCodes2>
__global__ void HQThirdStageL2Distances(Tensor<TVec, 2, true> queries,
                                        Tensor<int, 3, true> indices,
                                        const void** listCodes1,
                                        const void** listCodes2,
                                        Tensor<TVec, 4, true> codewords1,
                                        Tensor<TVec, 4, true> codewords2,
                                        int imiSize,
                                        Tensor<T, 2, true> distances) {
  extern __shared__ char smemByte[]; // #warps * RowTileSize elements
  T* smem = (T*) smemByte;

  int numWarps = utils::divUp(blockDim.x, kWarpSize);
  int laneId = getLaneId();
  int warpId = threadIdx.x / kWarpSize;

  int qid = blockIdx.y;

  bool lastRowTile = (blockIdx.x == (gridDim.x - 1));
  int rowStart = RowTileSize * blockIdx.x;
  T rowNorm[RowTileSize];

  if (lastRowTile) {
    // We are handling the very end of the input matrix rows
    for (int row = 0; row < indices.getSize(2) - rowStart; ++row) {
      //printf("indices[0][%d][%d]: %d\n", qid, rowStart + row, (int)indices[0][qid][rowStart + row]);
      int imiId[2] = {indices[0][qid][rowStart + row], indices[1][qid][rowStart + row]};
      int fineId = indices[2][qid][rowStart + row];
      int listId = imiId[0] * imiSize + imiId[1];

      if (NormLoop) {
        rowNorm[0] = 0;

        for (int col = threadIdx.x;
             col < queries.getSize(1); col += blockDim.x) {
          T val = HQThirdStageL2DistancesOneCol<numCodes2, T>(queries, listCodes1, listCodes2, codewords1, codewords2, imiId, fineId, listId, col);
          rowNorm[0] += val;
        }
      } else {
        T val = HQThirdStageL2DistancesOneCol<numCodes2, T>(queries, listCodes1, listCodes2, codewords1, codewords2, imiId, fineId, listId, threadIdx.x);
        rowNorm[0] = val;
      }

      rowNorm[0] = warpReduceAllSum(rowNorm[0]);
      if (laneId == 0) {
        smem[row * numWarps + warpId] = rowNorm[0];
      }
    }
  } else {
    // We are guaranteed that all RowTileSize rows are available in
    // [rowStart, rowStart + RowTileSize)

    if (NormLoop) {
      // A single block of threads is not big enough to span each
      // vector

#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        //printf("indices[0][%d][%d]: %d\n", qid, rowStart + row, (int)indices[0][qid][rowStart + row]);
        int imiId[2] = {indices[0][qid][rowStart + row], indices[1][qid][rowStart + row]};
        int fineId = indices[2][qid][rowStart + row];
        int listId = imiId[0] * imiSize + imiId[1];

        rowNorm[row] = 0;

        for (int col = threadIdx.x;
             col < queries.getSize(1); col += blockDim.x) {
          T val = HQThirdStageL2DistancesOneCol<numCodes2, T>(queries, listCodes1, listCodes2, codewords1, codewords2, imiId, fineId, listId, col);
          rowNorm[row] += val;
        }
      }
    } else {
      // A block of threads is the exact size of the vector

#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        //printf("indices[0][%d][%d]: %d\n", qid, rowStart + row, (int)indices[0][qid][rowStart + row]);
        int imiId[2] = {indices[0][qid][rowStart + row], indices[1][qid][rowStart + row]};
        int fineId = indices[2][qid][rowStart + row];
        int listId = imiId[0] * imiSize + imiId[1];

        T val = HQThirdStageL2DistancesOneCol<numCodes2, T>(queries, listCodes1, listCodes2, codewords1, codewords2, imiId, fineId, listId, threadIdx.x);
        rowNorm[row] = val;
      }
    }

    // Sum up all parts in each warp
#pragma unroll
    for (int row = 0; row < RowTileSize; ++row) {
      rowNorm[row] = warpReduceAllSum(rowNorm[row]);
    }

    if (laneId == 0) {
#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        smem[row * numWarps + warpId] = rowNorm[row];
      }
    }
  }

  __syncthreads();

  // Sum across warps
  if (warpId == 0) {
#pragma unroll
    for (int row = 0; row < RowTileSize; ++row) {
      rowNorm[row] = laneId < numWarps ?
                              smem[row * numWarps + laneId] : Math<T>::zero();
    }

#pragma unroll
    for (int row = 0; row < RowTileSize; ++row) {
      rowNorm[row] = warpReduceAllSum(rowNorm[row]);
    }

    // Write out answer
    if (laneId == 0) {
#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        int outCol = rowStart + row;

        if (lastRowTile) {
          if (outCol < distances.getSize(1)) {
            distances[qid][outCol] =
              NormSquared ? rowNorm[row] :
              ConvertTo<T>::to(
                sqrtf(ConvertTo<float>::to(rowNorm[row])));
          }
        } else {
          distances[qid][outCol] =
            NormSquared ? rowNorm[row] :
            ConvertTo<T>::to(
              sqrtf(ConvertTo<float>::to(rowNorm[row])));
        }
      }
    }
  }
}

void runHQThirdStageL2Distances(const Tensor<float, 2, true>& queries,
                                const Tensor<int, 3, true>& indices,
                                const void** listCodes1,
                                const void** listCodes2,
                                const Tensor<float, 4, true>& codewords1,
                                const Tensor<float, 4, true>& codewords2,
                                int imiSize,
                                int numCodes2,
                                Tensor<float, 2, true>& distances,
                                bool normSquared,
                                hipStream_t stream) {
  int64_t maxThreads = (int64_t) getMaxThreadsCurrentDevice();
  constexpr int rowTileSize = 8;

#define RUN_L2(TYPE_T, TYPE_TVEC, QUERIES, CODEWORDS1, CODEWORDS2, NUMCODES2)                                \
  do {                                                                  \
    if (normLoop) {                                                     \
      if (normSquared) {                                                \
        HQThirdStageL2Distances<TYPE_T, TYPE_TVEC, rowTileSize, true, true, NUMCODES2>      \
          <<<grid, block, smem, stream>>>(QUERIES, indices, listCodes1, listCodes2, CODEWORDS1, CODEWORDS2, imiSize, distances);               \
      } else {                                                          \
        HQThirdStageL2Distances<TYPE_T, TYPE_TVEC, rowTileSize, true, false, NUMCODES2>     \
          <<<grid, block, smem, stream>>>(QUERIES, indices, listCodes1, listCodes2, CODEWORDS1, CODEWORDS2, imiSize, distances);               \
      }                                                                 \
    } else {                                                            \
      if (normSquared) {                                                \
        HQThirdStageL2Distances<TYPE_T, TYPE_TVEC, rowTileSize, false, true, NUMCODES2>     \
          <<<grid, block, smem, stream>>>(QUERIES, indices, listCodes1, listCodes2, CODEWORDS1, CODEWORDS2, imiSize, distances);               \
      } else {                                                          \
        HQThirdStageL2Distances<TYPE_T, TYPE_TVEC, rowTileSize, false, false, NUMCODES2>    \
          <<<grid, block, smem, stream>>>(QUERIES, indices, listCodes1, listCodes2, CODEWORDS1, CODEWORDS2, imiSize, distances);               \
      }                                                                 \
    }                                                                   \
  } while (0)

  if (queries.canCastResize<float4>() && codewords1.canCastResize<float4>() && codewords2.canCastResize<float4>()) {
    // Can load using the vectorized type
    auto queriesV = queries.castResize<float4>();
    auto codewords1V = codewords1.castResize<float4>();
    auto codewords2V = codewords2.castResize<float4>();

    auto dim = queriesV.getSize(1);
    bool normLoop = dim > maxThreads;
    auto numThreads = min(dim, (int)maxThreads);

    auto grid = dim3(utils::divUp(indices.getSize(2), rowTileSize), queriesV.getSize(0));
    auto block = dim3(numThreads);

    auto smem = sizeof(float) * rowTileSize * utils::divUp(numThreads, kWarpSize);

    switch (numCodes2) {
      case 2: {
        RUN_L2(float, float4, queriesV, codewords1V, codewords2V, 2);
        break;
      }
      case 4: {
        RUN_L2(float, float4, queriesV, codewords1V, codewords2V, 4);
        break;
      }
      case 6: {
        RUN_L2(float, float4, queriesV, codewords1V, codewords2V, 6);
        break;
      }
      case 8: {
        RUN_L2(float, float4, queriesV, codewords1V, codewords2V, 8);
        break;
      }
      default: {
        FAISS_ASSERT_MSG(false, "This number of code 2 is not supported");
      }
    }
  } else {
    // Can't load using the vectorized type

    auto dim = queries.getSize(1);
    bool normLoop = dim > maxThreads;
    auto numThreads = min((int64_t)dim, maxThreads);

    auto grid = dim3(utils::divUp(indices.getSize(2), rowTileSize), queries.getSize(0));
    auto block = dim3(numThreads);

    auto smem = sizeof(float) * rowTileSize * utils::divUp(numThreads, kWarpSize);

    switch (numCodes2) {
      case 2: {
        RUN_L2(float, float, queries, codewords1, codewords2, 2);
        break;
      }
      case 4: {
        RUN_L2(float, float, queries, codewords1, codewords2, 4);
        break;
      }
      case 6: {
        RUN_L2(float, float, queries, codewords1, codewords2, 6);
        break;
      }
      case 8: {
        RUN_L2(float, float, queries, codewords1, codewords2, 8);
        break;
      }
      default: {
        FAISS_ASSERT_MSG(false, "This number of code 2 is not supported");
      }
    }
  }

#undef RUN_L2

  CUDA_TEST_ERROR();
}

} } // namespace
